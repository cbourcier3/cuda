#include "hip/hip_runtime.h"

#include <stdio.h>
#include "omp_repair.h"
#include <hip/hip_runtime.h>
static long num_steps = 1000000; // 100 millions
double step;

#if __CUDA_ARCH__ < 600
__device__ double atomicAdd2(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

__global__ void cal_pi(long num_steps, double step, double *sum)
{
		int i;
		double x;
		double local;
		local=0.0;
	for (i=1;i<= num_steps; i++){
		x = (i-0.5)*step;
		local += 4.0/(1.0+x*x);
	}
		atomicAdd2(sum, local);
}

int main ()
{
	  double pi, sum = 0.0;
	  double start_time, run_time;
	  step = 1.0/(double) num_steps;
	  start_time = omp_get_wtime();
		double *dev_sum;
    // capture the start time
    hipEvent_t     start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );
//
		hipMalloc((void **)&dev_sum, sizeof(double));
		hipMemset(dev_sum, 0, sizeof(double));
		cal_pi<<<1,1>>>(num_steps,step,dev_sum);
		hipMemcpy(&sum, dev_sum, sizeof(double), hipMemcpyDeviceToHost);

	  pi = step * sum;
    // get stop time, and display the timing results
		    hipEventRecord( stop, 0 );
		    hipEventSynchronize( stop );
		    float   elapsedTime;
		    hipEventElapsedTime( &elapsedTime,
		                                        start, stop );
		    printf( "Time to compute :  %3.1f ms\n", elapsedTime );
		    hipEventDestroy( start );
		    hipEventDestroy( stop );	
	  run_time = omp_get_wtime() - start_time;
	  printf("\n pi with %ld steps is %lf in %lf seconds\n ",num_steps,pi,run_time);
}
